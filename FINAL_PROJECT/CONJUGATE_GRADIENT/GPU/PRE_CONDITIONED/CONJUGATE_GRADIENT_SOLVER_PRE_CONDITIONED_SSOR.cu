/* Use SSOR pre-condition */ 

#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void FPRINTF(FILE*, int N, double, double*);
void PRE_CONDITION_SSOR(int N, double, double, double, double*, double*);
double EVALUATE_ERROR(int, int, double*);

__global__ void INITIALIZE(int N, double dx, double* rho, double* field, double* field_analytic)
{
	int idx_x = threadIdx.x + blockIdx.x*blockDim.x;
	int idx_y = threadIdx.y + blockIdx.y*blockDim.y;
	int idx = idx_x + idx_y*N;

	double x = idx_x*dx;
	double y = idx_y*dx;

	field_analytic[idx] = x*(1.-x)*y*(1.-y)*exp(x-y);
		
	if (idx_x!=0&&idx_x!=N-1&&idx_y!=0&&idx_y!=N-1)
	{
		field[idx] = 0.0;
		rho[idx] = (2.*x*(y-1)*(y-2.*x+x*y+2)*exp(x-y))*dx*dx;	// Notice that rho has been times by dx^2!!
	}
	else
	{
		field[idx] = field_analytic[idx];
		rho[idx] = 0.0;
	}
}

__global__ void EVALUATE_ERROR_BLOCK(int N, double* rho, double* field, double* error_block)
{
	extern __shared__ double sm[];
	int idx_x = threadIdx.x + blockIdx.x*blockDim.x;
	int idx_y = threadIdx.y + blockIdx.y*blockDim.y;
	int idx = idx_x + N*idx_y;
	int idx_sm = threadIdx.x + blockDim.x*threadIdx.y;

	if (idx_x!=0&&idx_x!=N-1&&idx_y!=0&&idx_y!=N-1)
	{
		int L = idx_x-1 + idx_y*N;
		int R = idx_x+1 + idx_y*N;
		int U = idx_x + (idx_y+1)*N;
		int D = idx_x + (idx_y-1)*N;
		sm[idx_sm] = pow((field[L]+field[R]+field[U]+field[D]-4.*field[idx])-rho[idx], 2.);
	}
	else
		sm[idx_sm] = 0.0;
	__syncthreads();

	for (int shift=blockDim.x*blockDim.y/2; shift>0; shift/=2)
	{
		if (idx_sm<shift)
			sm[idx_sm] += sm[idx_sm+shift];
		__syncthreads();
	}
	if (idx_sm==0)
		error_block[blockIdx.x+blockIdx.y*gridDim.x] = sm[0];
//	printf("%d\t%.4f\n", blockIdx.x+gridDim.x*blockIdx.y, sm[0]);
}

__global__ void LAPLACIAN(int N, double dx, double photon_mass, double* p, double* A_p)
{
	int idx_x = threadIdx.x + blockDim.x*blockIdx.x;
	int idx_y = threadIdx.y + blockDim.y*blockIdx.y;
	int idx = idx_x + N*idx_y;
	
	if (idx_x!=0&&idx_x!=N-1&&idx_y!=0&&idx_y!=N-1)
	{
		int L = idx_x-1 + idx_y*N;
		int R = idx_x+1 + idx_y*N;
		int U = idx_x + (idx_y+1)*N;
		int D = idx_x + (idx_y-1)*N;

		A_p[idx] = (p[L]+p[R]+p[U]+p[D]-(4.+pow(photon_mass*dx,2.))*p[idx]);
//		printf("%d\t%.4f\n", idx, A_p[idx]);
	}
	else
		A_p[idx] = 0.0;
}

__global__ void DAXPY(int N, double c, double *A, double *B)
{
	int idx_x = threadIdx.x + blockDim.x*blockIdx.x;
	int idx_y = threadIdx.y + blockDim.y*blockIdx.y;
	int idx = idx_x + N*idx_y;

	A[idx] = c*A[idx] + B[idx];
}

int main(void)
{
	int N, N_block, display_interval, tpb_x, tpb_y, bpg_x, bpg_y;
	float preparation_time, computation_time, total_time;
	double photon_mass, omega, dx, criteria;
	double alpha, beta;
	long iter, iter_max;
	double *field, *rho, *r, *r_prime, *p, *A_p, *field_analytic, *error_block;
	size_t size_lattice, size_sm;
	hipEvent_t start, stop;
	FILE* output_field, *output_rho;
	printf("Solve the Poission problem using CG with SSOR precondition by GPU.\n\n");
	printf("Enter the latttice size (N,N) .");
	scanf("%d", &N);
	printf("The lattice size is (%d,%d).\n", N, N);
	printf("Set the photon mass.\n");
	scanf("%lf", &photon_mass);
	printf("The photon mass is %.4e .\n", photon_mass);
	printf("Set the value of omega.\n");
	scanf("%lf",&omega);
	printf("The value of omega is %.4e .\n", omega);
	printf("Set the maximum iteration times.\n");
	scanf("%ld", &iter_max);
	printf("The maximum iteration times is %ld .\n", iter_max);
	printf("Set the stopping criteria.\n");
	scanf("%lf", &criteria);
	printf("The stopping criteria is %.4e .\n", criteria);
	printf("Set the display interval during iterations.\n");
	scanf("%d", &display_interval);
	printf("The display interval is set to be %d .\n", display_interval);
	printf("Set the GPU threads per block (tx,ty). (N must be divisible by tx and N must be divisible by ty)\n");
	scanf("%d %d", &tpb_x, &tpb_y);
	if (N%tpb_x!=0)
	{
		printf("N is not divisible by tx! Exit!\n");
		return EXIT_FAILURE;
	}
	else if (N%tpb_y!=0)
	{
		printf("N is not divisible by ty! Exit!\n");
		return EXIT_FAILURE;
	}
	else
	{
		printf("Threads per block for GPU is (%d,%d) .\n", tpb_x, tpb_y);
		printf("The block per grid will be set automatically.");
		bpg_x = N/tpb_x;
		bpg_y = N/tpb_y;
		printf("Blocks per grid for GPU is (%d,%d) .\n", bpg_x, bpg_y);
	}
	printf("\n");

	printf("Start Preparation...\n");
	hipSetDevice(0);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	dx = 1./(N-1);	
	N_block = bpg_x*bpg_y;
	size_lattice = N*N*sizeof(double);
	size_sm = tpb_x*tpb_y*sizeof(double);
	output_field = fopen("analytical_field_distribution_CG_precondition_SSOR.txt","w");
	output_rho = fopen("charge_distribution_CG_precondition_SSOR.txt","w");

	dim3 tpb(tpb_x,tpb_y);
	dim3 bpg(bpg_x,bpg_y);
	hipblasMath_t mode = HIPBLAS_TENSOR_OP_MATH;
    hipblasPointerMode_t mode_pt = HIPBLAS_POINTER_MODE_HOST;
	hipblasHandle_t handle;

	hipblasCreate(&handle);
	hipblasSetMathMode(handle, mode);
    hipblasSetPointerMode(handle, mode_pt);

	hipEventRecord(start,0);
	hipMallocManaged(&field, size_lattice);
	hipMallocManaged(&r, size_lattice);
	hipMallocManaged(&r_prime, size_lattice);
	hipMallocManaged(&p, size_lattice);
	hipMallocManaged(&A_p, size_lattice);
	hipMallocManaged(&field_analytic, size_lattice);
	hipMallocManaged(&rho, size_lattice);
	hipMallocManaged(&error_block, N_block*sizeof(double));

	INITIALIZE<<<bpg,tpb>>>(N, dx, rho, field, field_analytic);
	EVALUATE_ERROR_BLOCK<<<bpg,tpb,size_sm>>>(N, rho, field, error_block);
	double norm;
	hipblasDdot(handle, N*N, rho, 1, rho, 1, &norm);
	norm = sqrt(norm);
	
	hipDeviceSynchronize();
	hipMemcpy(r, rho, size_lattice, hipMemcpyDeviceToDevice);
	
	FPRINTF(output_field, N, 1., field_analytic);
	FPRINTF(output_rho, N, pow(dx,-2.), rho);

	printf("Preparation ends.\n");
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&preparation_time, start, stop);
	printf("Total preparation time is %.4f ms.\n\n", preparation_time);

	hipEventRecord(start,0);	
	double error = EVALUATE_ERROR(N, N_block, error_block); 
	double temp;

	printf("Starts computation with error = %.8e...\n", sqrt(error)/norm);
	iter = 0;
	PRE_CONDITION_SSOR(N, dx, photon_mass, omega, r, r_prime);

//	for (int i=0; i<N*N; i++)
//		printf("%.4f\n", r_prime[i]);

	hipMemcpy(p, r_prime, size_lattice, hipMemcpyDeviceToDevice);
	
	while (sqrt(error)/norm>criteria&&iter<iter_max)
	{
		LAPLACIAN<<<bpg,tpb>>>(N, dx, photon_mass, p, A_p);
		hipblasDdot(handle, N*N, p, 1, A_p, 1, &temp);
		hipblasDdot(handle, N*N, r, 1, r_prime, 1, &beta);
		alpha = beta/temp;
		temp = -alpha;
		hipblasDaxpy(handle, N*N, &temp, A_p, 1, r, 1);
		hipblasDaxpy(handle, N*N, &alpha, p, 1, field, 1);
		hipDeviceSynchronize();
		PRE_CONDITION_SSOR(N, dx, photon_mass, omega, r, r_prime);
		hipblasDdot(handle, N*N, r, 1, r_prime, 1, &temp);
		beta = temp/beta;
//		printf("%.4f\t%.4f\n", alpha, beta);
		DAXPY<<<bpg,tpb>>>(N, beta, p, r_prime);
		hipblasDdot(handle, N*N, r, 1, r, 1, &error);
		iter += 1;
		if (iter%display_interval==0)
			printf("Iteration = %ld , error = %.8e .\n", iter, sqrt(error)/norm);
	}
  
	output_field = fopen("simulated_field_distribution_GPU_CG_precondition_SSOR.txt","w");
	FPRINTF(output_field, N, 1., field);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&computation_time, start, stop);
	printf("Computation time is %.4f ms.\n", computation_time);
	total_time = preparation_time + computation_time;
	printf("Total iteration is %ld ; total time is %.4f ms.\n", iter, total_time);

	hipFree(field);
	hipFree(r);
	hipFree(r_prime);
	hipFree(p);
	hipFree(A_p);
	hipFree(field_analytic);
	hipFree(rho);
	hipFree(error_block);
	hipblasDestroy(handle);
	fclose(output_field);
	fclose(output_rho);
	return EXIT_SUCCESS;
}

double EVALUATE_ERROR(int N, int N_block, double* error_block)
{
	double error = 0.0;
	for (int i=0; i<N_block; i++)
		error += error_block[i];
	return error;
}

void PRE_CONDITION_SSOR(int N, double dx, double photon_mass, double omega, double* r, double* r_prime)
{
    for (int idx=0; idx<N*N; idx++)
    {
        int idx_x = idx%N;
        int idx_y = idx/N;
        r_prime[idx] = r[idx]*omega*(2.-omega);
        if ( idx_x!=0 && idx_x!=N-1 && idx_y!=0 && idx_y!=N-1 )
        {
            if (idx_x>1&&idx_y>1)
                r_prime[idx] = (-r_prime[idx]+omega*(r_prime[idx-1]+r_prime[idx-N]))/(4.+pow(photon_mass*dx,2.));
            else if (idx_x>1)
                r_prime[idx] = (-r_prime[idx]+omega*r_prime[idx-1])/(4.+pow(photon_mass*dx,2.));
            else if (idx_y>1)
                r_prime[idx] = (-r_prime[idx]+omega*r_prime[idx-N])/(4.+pow(photon_mass*dx,2.));
            else
                r_prime[idx] = -r_prime[idx]/(4.+pow(photon_mass*dx,2.));
        }
    }
    for (int idx=N*N-1; idx>=0; idx--)
    {
        int idx_x = idx%N;
        int idx_y = idx/N;
        if ( idx_x!=0 && idx_x!=N-1 && idx_y!=0 && idx_y!=N-1 )
        {
            r_prime[idx] *= -pow(photon_mass*dx,2.) - 4.;
            if (idx_x<N-2&&idx_y<N-2)
                r_prime[idx] = -(r_prime[idx]-omega*(r_prime[idx+1]+r_prime[idx+N]))/(4.+pow(photon_mass*dx,2.));
            else if (idx_x<N-2)
                r_prime[idx] = -(r_prime[idx]-omega*r_prime[idx+1])/(4.+pow(photon_mass*dx,2.));
            else if (idx_y<N-2)
                r_prime[idx] = -(r_prime[idx]-omega*r_prime[idx+N])/(4.+pow(photon_mass*dx,2.));
            else
                r_prime[idx] = -r_prime[idx]/(4.+pow(photon_mass*dx,2.));
        }
    }
}

void FPRINTF(FILE *output_file, int N, double scale, double *array)
{
	for (int j=0; j<N; j++)
	{
		for (int i=0; i<N; i++)
			fprintf(output_file, "%.8e\t", scale*array[i+j*N]);
		fprintf(output_file, "\n");
	}
}
