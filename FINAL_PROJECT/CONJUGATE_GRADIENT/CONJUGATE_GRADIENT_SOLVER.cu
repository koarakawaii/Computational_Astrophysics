#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void FPRINTF(FILE*, int N, double, double*);
double EVALUATE_ERROR(int, int, double*);

__global__ void INITIALIZE(int N, double dx, double* rho, double* field, double* field_analytic, double* error_block)
{
	extern __shared__ double sm[];
	int idx_x = threadIdx.x + blockIdx.x*blockDim.x;
	int idx_y = threadIdx.y + blockIdx.y*blockDim.y;
	int idx = idx_x + idx_y*N;
	int idx_sm = threadIdx.x + blockDim.x*threadIdx.y;

	double x = idx_x*dx;
	double y = idx_y*dx;

	field_analytic[idx] = x*(1.-x)*y*(1.-y)*exp(x-y);
		
	if (idx_x!=0&&idx_x!=N-1&&idx_y!=0&&idx_y!=N-1)
	{
		int L = idx_x-1 + idx_y*N;
		int R = idx_x+1 + idx_y*N;
		int U = idx_x + (idx_y+1)*N;
		int D = idx_x + (idx_y-1)*N;
		field[idx] = 0.0;
		rho[idx] = (2.*x*(y-1)*(y-2.*x+x*y+2)*exp(x-y))*dx*dx;	// Notice that rho has been times by dx^2!!
		sm[idx_sm] = pow((field[L]+field[R]+field[U]+field[D]-4.*field[idx])-rho[idx], 2.);
	}
	else
	{
		field[idx] = field_analytic[idx];
		rho[idx] = 0.0;
		sm[idx_sm] = 0.0;
	}
	__syncthreads();
//	printf("%d\t%d\t%.4f\n", idx, idx_sm, sm[idx_sm]);
	
	for (int shift=blockDim.x*blockDim.y/2; shift>0; shift/=2)
	{
		if (idx_sm<shift)
			sm[idx_sm] += sm[idx_sm+shift];
		__syncthreads();
	}
	if (idx_sm==0)
		error_block[blockIdx.x+blockIdx.y*gridDim.x] = sm[0];
//	printf("%d\t%.4f\n", blockIdx.x+gridDim.x*blockIdx.y, sm[0]);
}

__global__ void LAPLACIAN(int N, double dx, double photon_mass, double* p, double* A_p)
{
	int idx_x = threadIdx.x + blockDim.x*blockIdx.x;
	int idx_y = threadIdx.y + blockDim.y*blockIdx.y;
	int idx = idx_x + N*idx_y;
	
	if (idx_x!=0&&idx_x!=N-1&&idx_y!=0&&idx_y!=N-1)
	{
		int L = idx_x-1 + idx_y*N;
		int R = idx_x+1 + idx_y*N;
		int U = idx_x + (idx_y+1)*N;
		int D = idx_x + (idx_y-1)*N;

		A_p[idx] = (p[L]+p[R]+p[U]+p[D]-(4.-pow(photon_mass*dx,2.))*p[idx]);
//		printf("%d\t%.4f\n", idx, A_p[idx]);
	}
	else
		A_p[idx] = 0.0;
}

__global__ void DAXPY(int N, double c, double *A, double *B)
{
	int idx_x = threadIdx.x + blockDim.x*blockIdx.x;
	int idx_y = threadIdx.y + blockDim.y*blockIdx.y;
	int idx = idx_x + N*idx_y;

	A[idx] = c*A[idx] + B[idx];
}

int main(void)
{
	int N, N_block, display_interval, tpb_x, tpb_y, bpg_x, bpg_y;
	float preparation_time, computation_time, total_time;
	double photon_mass, dx, criteria;
	double alpha, beta;
	long iter, iter_max;
	double *field, *rho, *r, *p, *A_p, *field_analytic, *error_block;
	size_t size_lattice, size_sm;
	hipEvent_t start, stop;
	FILE* output_field, *output_rho;
	printf("Solve the Poission problem using SOR by OpenMP.\n\n");
	printf("Enter the latttice size (N,N) (N must be divisible by 2).");
	scanf("%d", &N);
	printf("The lattice size is (%d,%d).\n", N, N);
	printf("Set the photon mass.\n");
	scanf("%lf", &photon_mass);
	printf("The photon mass is %.4e .\n", photon_mass);
	printf("Set the maximum iteration times.\n");
	scanf("%ld", &iter_max);
	printf("The maximum iteration times is %ld .\n", iter_max);
	printf("Set the stopping criteria.\n");
	scanf("%lf", &criteria);
	printf("The stopping criteria is %.4e .\n", criteria);
	printf("Set the display interval during iterations.\n");
	scanf("%d", &display_interval);
	printf("The display interval is set to be %d .\n", display_interval);
	printf("Set the GPU threads per block (tx,ty). (N must be divisible by tx and N must be divisible by N)\n");
	scanf("%d %d", &tpb_x, &tpb_y);
	if (N%tpb_x!=0)
	{
		printf("N is not divisible by tx! Exit!\n");
		return EXIT_FAILURE;
	}
	else if (N%tpb_y!=0)
	{
		printf("N is not divisible by ty! Exit!\n");
		return EXIT_FAILURE;
	}
	else
	{
		printf("Threads per block for GPU is (%d,%d) .\n", tpb_x, tpb_y);
		printf("The block per grid will be set automatically.");
		bpg_x = N/tpb_x;
		bpg_y = N/tpb_y;
		printf("Blocks per grid for GPU is (%d,%d) .\n", bpg_x, bpg_y);
	}
	printf("\n");

	printf("Start Preparation...\n");
	dx = 1./(N-1);	
	N_block = (N/tpb_x)*(N/tpb_y);
	size_lattice = N*N*sizeof(double);
	size_sm = tpb_x*tpb_y*sizeof(double);
	output_field = fopen("analytical_field_distribution_CG.txt","w");
	output_rho = fopen("charge_distribution_CG.txt","w");

	hipSetDevice(0);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	dim3 tpb(tpb_x,tpb_y);
	dim3 bpg(bpg_x,bpg_y);
	hipblasMath_t mode = HIPBLAS_TENSOR_OP_MATH;
    hipblasPointerMode_t mode_pt = HIPBLAS_POINTER_MODE_HOST;
	hipblasHandle_t handle;

	hipblasCreate(&handle);
	hipblasSetMathMode(handle, mode);
    hipblasSetPointerMode(handle, mode_pt);

	hipEventRecord(start,0);
	hipMallocManaged(&field, size_lattice);
	hipMallocManaged(&r, size_lattice);
	hipMallocManaged(&p, size_lattice);
	hipMallocManaged(&A_p, size_lattice);
	hipMallocManaged(&field_analytic, size_lattice);
	hipMallocManaged(&rho, size_lattice);
	hipMallocManaged(&error_block, N_block*sizeof(double));

	INITIALIZE<<<bpg,tpb,size_sm>>>(N, dx, rho, field, field_analytic, error_block);
	hipDeviceSynchronize();
	hipMemcpy(r, rho, size_lattice, hipMemcpyDeviceToDevice);
	hipMemcpy(p, rho, size_lattice, hipMemcpyDeviceToDevice);
	
	FPRINTF(output_field, N, 1., field_analytic);
	FPRINTF(output_rho, N, pow(dx,-2.), rho);
	hipEventRecord(start,0);

	printf("Preparation ends.\n");
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&preparation_time, start, stop);
	printf("Total preparation time is %.4f ms.\n\n", preparation_time);

	hipEventRecord(start,0);	
	double error = EVALUATE_ERROR(N, N_block, error_block); 
	double temp;

	printf("Starts computation with error = %.8e...\n", error);
	iter = 0;
	
	while (sqrt(error)/(double)(N-2)>criteria&&iter<iter_max)
	{
		LAPLACIAN<<<bpg,tpb>>>(N, dx, photon_mass, p, A_p);
		hipblasDdot(handle, N*N, p, 1, A_p, 1, &temp);
		alpha = error/temp;
		temp = -alpha;
		hipblasDaxpy(handle, N*N, &temp, A_p, 1, r, 1);
		hipblasDaxpy(handle, N*N, &alpha, p, 1, field, 1);
		hipblasDdot(handle, N*N, r, 1, r, 1, &temp);
		beta = temp/error;
//		printf("%.4f\t%.4f\n", alpha, beta);
		DAXPY<<<bpg,tpb>>>(N, beta, p, r);
		error = temp;
		iter += 1;
		if (iter%display_interval==0)
			printf("Iteration = %ld , error = %.8e .\n", iter, sqrt(error)/(double)(N-2));
	}
  
	output_field = fopen("simulated_field_distribution_GPU_CG.txt","w");
	FPRINTF(output_field, N, 1., field);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&computation_time, start, stop);
	printf("Computation time is %.4f ms.\n", computation_time);
	total_time = preparation_time + computation_time;
	printf("Total iteration is %ld ; total time is %.4f ms.\n", iter, total_time);

	hipFree(field);
	hipFree(r);
	hipFree(p);
	hipFree(field_analytic);
	hipFree(rho);
	hipFree(error_block);
	hipblasDestroy(handle);
	fclose(output_field);
	fclose(output_rho);
	return EXIT_SUCCESS;
}

double EVALUATE_ERROR(int N, int N_block, double* error_block)
{
	double error = 0.0;
	for (int i=0; i<N_block; i++)
		error += error_block[i];
	return error;
}

void FPRINTF(FILE *output_file, int N, double scale, double *array)
{
	for (int j=0; j<N; j++)
	{
		for (int i=0; i<N; i++)
			fprintf(output_file, "%.8e\t", scale*array[i+j*N]);
		fprintf(output_file, "\n");
	}
}
